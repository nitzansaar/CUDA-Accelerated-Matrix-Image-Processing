#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

#define TILE_WIDTH 16

// -----------------------
// Shared-memory tiled kernel
// -----------------------
__global__ void matrixMultiplyTiled(float *A, float *B, float *C, int N) {
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    float Pvalue = 0.0f;

    for (int m = 0; m < (N + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        if (Row < N && (m*TILE_WIDTH + tx) < N)
            ds_A[ty][tx] = A[Row * N + m*TILE_WIDTH + tx];
        else
            ds_A[ty][tx] = 0.0f;

        if (Col < N && (m*TILE_WIDTH + ty) < N)
            ds_B[ty][tx] = B[(m*TILE_WIDTH + ty) * N + Col];
        else
            ds_B[ty][tx] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k)
            Pvalue += ds_A[ty][k] * ds_B[k][tx];

        __syncthreads();
    }

    if (Row < N && Col < N)
        C[Row * N + Col] = Pvalue;
}

// -----------------------
// Exposed C-API function
// -----------------------
extern "C" void gpu_matrix_multiply(float *h_A, float *h_B, float *h_C, int N) {
    size_t size = N * N * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid((N + TILE_WIDTH - 1) / TILE_WIDTH,
                 (N + TILE_WIDTH - 1) / TILE_WIDTH);

    matrixMultiplyTiled<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
