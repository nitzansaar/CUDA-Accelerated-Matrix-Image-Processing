#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <time.h>

int main(int argc, char **argv) {
    int N = (argc > 1) ? atoi(argv[1]) : 1024;
    size_t size = N * N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize matrices with random values
    for (int i = 0; i < N * N; i++) {
        h_A[i] = rand() % 100 / 100.0f;
        h_B[i] = rand() % 100 / 100.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta  = 0.0f;

    // Start timing
    clock_t start = clock();

    // Perform matrix multiplication using cuBLAS
    // Note: matrices are row-major, so swap A and B
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                N, N, N,
                &alpha,
                d_B, N,   // B first
                d_A, N,   // then A
                &beta,
                d_C, N);

    hipDeviceSynchronize();
    clock_t end = clock();

    double elapsed = (double)(end - start) / CLOCKS_PER_SEC;
    printf("cuBLAS GPU execution time (N=%d): %f seconds\n", N, elapsed);

    // Cleanup
    hipblasDestroy(handle);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
