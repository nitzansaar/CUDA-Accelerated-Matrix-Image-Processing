#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define TILE_WIDTH 16

// ----------------------------
// Tiled CUDA Kernel
// ----------------------------
__global__ void matrixMultiplyTiled(float *A, float *B, float *C, int N) {
    __shared__ float ds_A[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_B[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0.0f;

    // Loop over tiles
    for (int m = 0; m < (N + TILE_WIDTH - 1) / TILE_WIDTH; ++m) {
        // Load sub-matrices into shared memory
        if (Row < N && (m * TILE_WIDTH + tx) < N)
            ds_A[ty][tx] = A[Row * N + m * TILE_WIDTH + tx];
        else
            ds_A[ty][tx] = 0.0f;

        if (Col < N && (m * TILE_WIDTH + ty) < N)
            ds_B[ty][tx] = B[(m * TILE_WIDTH + ty) * N + Col];
        else
            ds_B[ty][tx] = 0.0f;

        __syncthreads();

        // Multiply the two tiles
        for (int k = 0; k < TILE_WIDTH; ++k)
            Pvalue += ds_A[ty][k] * ds_B[k][tx];

        __syncthreads();
    }

    // Write back to global memory
    if (Row < N && Col < N)
        C[Row * N + Col] = Pvalue;
}

// ----------------------------
// Host Code
// ----------------------------
int main(int argc, char **argv) {
    int N = (argc > 1) ? atoi(argv[1]) : 1024;
    size_t size = N * N * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    // Initialize input matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = rand() % 100 / 100.0f;
        h_B[i] = rand() % 100 / 100.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);
    dim3 dimGrid((N + TILE_WIDTH - 1) / TILE_WIDTH,
                 (N + TILE_WIDTH - 1) / TILE_WIDTH);

    clock_t start = clock();
    matrixMultiplyTiled<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);
    hipDeviceSynchronize();
    clock_t end = clock();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    double elapsed = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Optimized (tiled) GPU execution time (N=%d): %f seconds\n", N, elapsed);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}
